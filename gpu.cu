#include "hip/hip_runtime.h"
#include "gpu.hpp"

extern utils::DebugStream dbg;

namespace {

__global__ void classify(unsigned *l,
                        const double *d, 
                        const double *c, 
                        const size_t d_sz,
                        const size_t c_sz,
                        const size_t dim)
{
  size_t tid = blockDim.x*blockIdx.x + threadIdx.x;

  if(d_sz <= tid) return;

  const double *point = &d[tid*dim];

  unsigned res = 0;
  double min_d = HUGE_VAL;
  for(size_t cl = 0; cl < c_sz; ++cl){
    double diff = 0;
    const double *cluster = &c[cl*dim];
    for(size_t di = 0; di < dim; ++di) 
      diff += (point[di] - cluster[di])*(point[di] - cluster[di]);
    if(diff < min_d){
      min_d = diff;
      res = cl;
    }
  }
  l[tid] = res;
}

__device__ double atomicAddDouble(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed,
                      __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);

  return __longlong_as_double(old);
}

__global__ void update(double *tmp_c, 
                      const double *d, 
                      const unsigned *l,
                      const size_t c_sz,
                      const size_t d_sz,
                      const size_t dim)
{
  extern __shared__ char sh_all[];

  // set the counter and centroids locations in the shared memory
  unsigned *npts = (unsigned *)(&sh_all[0]);
  double *cent = (double *)(sh_all + c_sz*sizeof(unsigned));

  // initialize the shared memory
  if(threadIdx.x == 0){ // thread 0 of all blocks initializes it's shared mem
    for(size_t i = 0; i < c_sz; ++i){
      npts[i] = 0; // # of pts belonging to ith cluster
    }
    for(size_t cid = 0; cid < c_sz; ++cid){
      for(size_t dimid = 0; dimid < dim; ++dimid){
        cent[cid*dim + dimid] = 0;
      }
    }
  }

  __syncthreads(); // wait all threads until the initialization completes

  size_t tid = blockIdx.x*blockDim.x + threadIdx.x;
  atomicAdd(&npts[l[tid]], 1);
  for(size_t dimid = 0; dimid < dim; ++dimid)
    atomicAddDouble(&cent[l[tid]*dim + dimid], d[tid*dim + dimid]);

  __syncthreads(); // After all threads accumulate

  if(threadIdx.x == 0){
    double *c = &tmp_c[blockIdx.x*c_sz*dim];
    for(size_t cid = 0; cid < c_sz; ++cid){
      for(size_t dimid = 0; dimid < dim; ++dimid){
        if(npts[cid])
          cent[cid*dim + dimid] /= npts[cid]; // div with num of pts in the cluster
        c[cid*dim + dimid] = cent[cid*dim + dimid]; // centroids
      }
    }
  }
}

// use with only one block
__global__ void reduce(double *c, const double *tmp_c, const unsigned c_sz, const unsigned dim){
  extern __shared__ double sh_c[];

  unsigned tid = threadIdx.x;

  for(unsigned cid = 0; cid < c_sz; ++cid){
    for(unsigned did = 0; did < dim; ++did){
      sh_c[threadIdx.x*c_sz*dim + cid*dim + did] = tmp_c[threadIdx.x*c_sz*dim + cid*dim + did];
    }
  }

  __syncthreads();
  
  // Perform reduction in shared memory
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      //sdata[tid] += sdata[tid + s];
      for(unsigned cid = 0; cid < c_sz; ++cid){
        for(unsigned did = 0; did < dim; ++did){
          sh_c[tid*c_sz*dim + c_sz*dim + did] += sh_c[(tid*c_sz*dim + s)*c_sz*dim + c_sz*dim + did];
        }
      }
    }
    __syncthreads();
  }

  // copy the centroid to global memory
  if(tid == 0){
    for(unsigned cid = 0; cid < c_sz; ++cid){
      for(unsigned did = 0; did < dim; ++did){
        sh_c[c_sz*dim +  did] += sh_c[c_sz*dim + did];
      }
    }
  }
}


}

namespace utils {

KmeansStrategyGpuGlobalBase::KmeansStrategyGpuGlobalBase(const size_t sz, const size_t k, const size_t dim)
  : d_sz_{sz}, c_sz_{k}, dim_{dim}
{
  gpuErrchk( hipMalloc((void**)&data_device_, sizeof(double)*dim*sz) );
  gpuErrchk( hipMalloc((void**)&c_device_, sizeof(double)*dim*k) );
  gpuErrchk( hipMalloc((void**)&old_c_device_, sizeof(double)*dim*k) );
  gpuErrchk( hipMalloc((void**)&labels_device_, sizeof(unsigned)*sz) );

  // temporary storage before the centroids are accumulated
  gpuErrchk( hipMalloc((void**)&tmp_c_device_, sizeof(double)*dim*k*Args::blocks_update) );

  hipEventCreate(&start);
  hipEventCreate(&stop);
}

void KmeansStrategyGpuGlobalBase::init(const double *d, const double *c, const size_t data_sz, const size_t c_sz) {
  float tm; 
  startGpuTimer();
  gpuErrchk( hipMemcpy(data_device_, d, data_sz, hipMemcpyHostToDevice) );
  tm = endGpuTimer();
  registerTime(KmeansStrategyGpuGlobalBase::MEMCPY, tm);
  if(Args::debug){
    std::vector<double> d_test(data_sz/sizeof(double), 0);
    gpuErrchk( hipMemcpy(&d_test[0], data_device_, data_sz, hipMemcpyDeviceToHost) );
    for(int i = 0; i < data_sz/sizeof(double); ++i) assert(d_test[i] == d[i]);
  }

  startGpuTimer();
  gpuErrchk( hipMemcpy(c_device_, c, c_sz, hipMemcpyHostToDevice) );
  tm = endGpuTimer();
  registerTime(KmeansStrategyGpuGlobalBase::MEMCPY, tm);
  if(Args::debug){
    std::vector<double> c_test(c_sz/sizeof(double), 0);
    gpuErrchk( hipMemcpy(&c_test[0], c_device_, c_sz, hipMemcpyDeviceToHost) );
    for(int i = 0; i < c_sz/sizeof(double); ++i) assert(c_test[i] == c[i]);
  }

  startGpuTimer();
  gpuErrchk( hipMemcpy(old_c_device_, c, c_sz, hipMemcpyHostToDevice) );
  tm = endGpuTimer();
  registerTime(KmeansStrategyGpuGlobalBase::MEMCPY, tm);
}

void KmeansStrategyGpuGlobalBase::collect(double *c, unsigned *l, const size_t c_sz, const size_t l_sz) {
  float tm; 
  startGpuTimer();
  gpuErrchk( hipMemcpy(c, c_device_, c_sz, hipMemcpyDeviceToHost) );
  tm = endGpuTimer();
  registerTime(KmeansStrategyGpuGlobalBase::MEMCPY, tm);

  startGpuTimer();
  gpuErrchk( hipMemcpy(l, labels_device_, d_sz_, hipMemcpyDeviceToHost) );
  tm = endGpuTimer();
  registerTime(KmeansStrategyGpuGlobalBase::MEMCPY, tm);
}

void KmeansStrategyGpuGlobalBase::swap() { std::swap(c_device_, old_c_device_); }

KmeansStrategyGpuGlobalBase::~KmeansStrategyGpuGlobalBase() {
  static const std::string event_names[_EVENT_TYPE_LEN] = {
    "MEMCPY", 
    "CLASSIFY", 
    "UPDATE", 
    "OTHERS", 
    };

  dbg << "GPU time: \n";
  for(const auto &[type, v]: event_times_){
    dbg << event_names[type] << ": ";
    float total = 0;
    for(const auto t: v){
      dbg << t << ' ';
      total += t;
    }
    dbg << '\n';
    dbg << "total = " << total << '\n';
  }

  gpuErrchk( hipFree(data_device_) );
  gpuErrchk( hipFree(c_device_) );
  gpuErrchk( hipFree(old_c_device_) );
  gpuErrchk( hipFree(labels_device_) );
  gpuErrchk( hipFree(tmp_c_device_) );

  hipEventDestroy(start);
  hipEventDestroy(stop); 
}

void KmeansStrategyGpuGlobalBase::getCentroids(double *host_c) {
  dbg << __func__ << '\n';
  startGpuTimer();
  gpuErrchk( hipMemcpy(host_c, c_device_, sizeof(double)*c_sz_*dim_, hipMemcpyDeviceToHost) );
  float tm = endGpuTimer();
  registerTime(KmeansStrategyGpuGlobalBase::MEMCPY, tm);
}

void KmeansStrategyGpuBaseline::findNearestCentroids() {
  const unsigned NTHREADS = Args::threads_classify;
  const unsigned NBLOCKS = Args::blocks_classify;

  dbg << "classify<<< " << NBLOCKS << ", " << NTHREADS << " >>>()";
  startGpuTimer();
  classify<<<NTHREADS, NBLOCKS>>>(labels_device_, 
                        data_device_, 
                        old_c_device_, 
                        d_sz_, 
                        c_sz_, 
                        dim_);   
  gpuErrchk( hipPeekAtLastError() );
  float tm = endGpuTimer();
  registerTime(KmeansStrategyGpuGlobalBase::CLASSIFY, tm);

  if(Args::debug){
    static std::unique_ptr<unsigned> labels {new unsigned[d_sz_]};
    hipMemcpy(labels.get(), labels_device_, sizeof(unsigned)*d_sz_, hipMemcpyDeviceToHost);
    dbg << "Labels: ";
    for(int i = 0; i < d_sz_; ++i) dbg << labels.get()[i] << ' ';
    dbg << '\n';
  }
}

void KmeansStrategyGpuBaseline::averageLabeledCentroids() 
{
  const unsigned NTHREADS = Args::threads_update;
  const unsigned NBLOCKS = Args::blocks_update;

  dbg <<  "update<<< " << NBLOCKS << ", " << NTHREADS << " >>>()";
  startGpuTimer();
  update<<<NBLOCKS, 
          NTHREADS, 
          sizeof(unsigned)*c_sz_ + sizeof(double)*c_sz_*dim_>>>
          (tmp_c_device_, data_device_, labels_device_,
          c_sz_, d_sz_, dim_);
  gpuErrchk( hipPeekAtLastError() );

  hipDeviceSynchronize();
  gpuErrchk( hipPeekAtLastError() );

  float tm = endGpuTimer();
  registerTime(KmeansStrategyGpuGlobalBase::UPDATE, tm);


  // one thread is enough 
  dbg <<  "reduce<<< " << 1 << ", " << NBLOCKS << " >>>()";
  startGpuTimer();
  reduce<<<1, NBLOCKS>>>(c_device_, tmp_c_device_, c_sz_, dim_);
  gpuErrchk( hipPeekAtLastError() );
  tm = endGpuTimer();
  registerTime(KmeansStrategyGpuGlobalBase::UPDATE, tm);

  if(Args::debug){
    DoubleCentroids tmp_c{c_sz_, dim_};
    gpuErrchk( hipMemcpy(tmp_c.ptr(), c_device_, c_sz_*dim_, hipMemcpyDeviceToHost) );
    dbg << "Updated Centroids: \n";
    print_centroids(dbg, tmp_c);
  }

}

/* KmeansGpu */
Labels KmeansGpu::fit(){

  dbg << __func__ << '\n';

  auto &c = KmeansBase<KmeansGpu>::c_;
  auto &old_c = KmeansBase<KmeansGpu>::old_c_;
  auto &d = KmeansBase<KmeansGpu>::d_;
  auto &solved = KmeansBase<KmeansGpu>::solved_;
  auto &iters = KmeansBase<KmeansGpu>::iters_;
  auto &max_iters = KmeansBase<KmeansGpu>::max_iters_;

  stgy_->init(d.ptr(), 
              c.ptr(), 
              sizeof(double)*d.size()*d.dim(), 
              sizeof(double)*c.size()*c.dim());

  Labels l(d.size(), 0);

  while(true){
    // labels is a mapping from each point in the dataset 
    // to the nearest (euclidean distance) centroid
    stgy_->findNearestCentroids();

    // the new centroids are the average 
    // of all the points that map to each 
    // centroid
    stgy_->averageLabeledCentroids();
    dbg << "ITER = " << iters << '\n';
    //print_centroids(dbg, c_);
    stgy_->getCentroids(c.ptr());
    if(++iters > max_iters || converged(c.ptr(), old_c.ptr(), c.size(), c.dim())) break;
    //done = ++iters_ > max_iters_;

    stgy_->swap();
    std::swap(c, old_c);
  }

  stgy_->collect(c.ptr(), 
                &l[0], 
                sizeof(double)*c.dim()*c.size(), 
                sizeof(unsigned)*d.size());

  solved = true;
  return l;
}

}
